#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// CUDA kernel for initializing a vector
__global__ void initializeVector(double *data, size_t size) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = static_cast<double>(idx);
    }
}

void runCudaTest(size_t N, std::vector<double> &hostData) {
    double *deviceData;
    hipMalloc(&deviceData, N * sizeof(double));
    initializeVector<<<(N + 255) / 256, 256>>>(deviceData, N);
    hipMemcpy(hostData.data(), deviceData, N * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(deviceData);
}
