#include <hip/hip_runtime.h>

// CUDA kernel for initializing a vector
__global__ void initializeVector(double *data, size_t size) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = static_cast<double>(idx);
    }
}
